#include "hip/hip_runtime.h"
#include <limits>
// #include "custom_cuda_layers.h"

#include <benchmark/benchmark.h>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime_api.h>
#include <torch/extension.h>

#include <cstdio>
#include <cstdlib>
#include <ctime>

#include "context.h"

namespace cg = cooperative_groups;

#define INPUT_TILE 1
#define INPUT_TILE1 1

// Input tile used in the gemm kernel v2
#define INPUT_TILE2_Q 8

#define INPUT_TILE2 8

#define MAX_REG_SIZE 20

// https://github.com/microsoft/DeepSpeed-internal/blob/inference-specialized-only/csrc/transformer/inference_specialized/includes/custom_cuda_layers.h#L11
#define WARP_SIZE 32
#define SMs 160
#define CACHLINE 128
#define MAX_REGISTERS 256

#define MAX_WARP_NUM 32
#define MAX_BLOCK_SUM 8

#define loop_unroll 4
#define loop_unroll_bits 2

#define inner_loop_unroll 4
#define inner_loop_unroll_bits 2

#define INT8WIDTH 2

#define MAX_QUANTIZE_GROUPING 1024

#define ACC_HALF true

inline __device__ float gelu(const float x) {
  float y = 0.5 * x *
            (1.0 + tanhf(0.7978845608028654 * x * (1.0 + 0.044715 * x * x)));
  return y;
}

void CheckCudaErrorAux(const char* file, unsigned line)

{
  hipError_t err = hipGetLastError();

  if (err == hipSuccess) return;

  std::cerr << hipGetErrorString(err) << "(" << err << ") at " << file << ":"
            << line

            << std::endl;

  throw std::runtime_error("CUDA ERROR!!!\n");
}

#define CUDA_CHECK_ERROR() CheckCudaErrorAux(__FILE__, __LINE__)

// https://github.com/microsoft/DeepSpeed-internal/blob/reyazda/fast-attn/csrc/transformer/inference_specialized/csrc/custom_gemm.cu#L43
__global__ void input_tiled_gemm_kernel_v2(
    __half* output, const __half* vals, const int8_t* weight,
    const __half* bias, unsigned hidden_dim, unsigned block_reduce,
    unsigned input_size, unsigned output_size, unsigned outputBlocks,
    unsigned blockStride, float* qscale, unsigned groups, __half* block_sums,
    unsigned merge_count = 1, unsigned quantization_stride = 1,
    bool add_gelu = false) {
  // #if __CUDA_ARCH__ >= 700
  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);

  unsigned int gid = threadIdx.x >> 5;
  unsigned int lane = threadIdx.x & 0x1f;

  int warp_num = blockDim.x >> 5;

  float2* output_cast = reinterpret_cast<float2*>(
      ((gridDim.x == outputBlocks) ? output : block_sums));
  const float4* vals_cast = reinterpret_cast<const float4*>(vals);
  const float2* qscale_cast = reinterpret_cast<const float2*>(qscale);
  const float4* weight_cast = reinterpret_cast<const float4*>(weight);

  output_cast += (unsigned)(blockIdx.x / outputBlocks) * (output_size);
  weight_cast += ((unsigned)(blockIdx.x / outputBlocks) * blockStride);
  vals_cast += (unsigned)(blockIdx.x / outputBlocks) * (hidden_dim >> 3);
  int output_size_quarter = output_size >> 2;
  // reading all the quantization scale into a small shared buffer
  __shared__ float2 shared_quantize_scale[MAX_QUANTIZE_GROUPING];

  __shared__ float2 partial_result[2 * MAX_WARP_NUM * (WARP_SIZE + 2)];
  if ((threadIdx.x << 1) < ((groups << merge_count)))
    shared_quantize_scale[threadIdx.x] = (qscale_cast[threadIdx.x]);
  __syncthreads();
  unsigned hidden_quarter = (hidden_dim >> 2);
  // for (int j = 0; j < input_size; j++)
  {
    float2 sum[2];
#pragma unroll
    for (int t = 0; t < 2; t++) {
      sum[t].x = 0.f;
      sum[t].y = 0.f;
    }

    {
      weight_cast += (gid << 3) * output_size_quarter +
                     (blockIdx.x % outputBlocks) * WARP_SIZE + lane;
      int col = (blockIdx.x % outputBlocks) * WARP_SIZE + lane;
      float4 weight_q[2];
      if (col < output_size) {
        weight_q[0] = weight_cast[0];
        weight_q[1] = weight_cast[output_size];
      }
      float4 val_h;
      val_h = vals_cast[gid];
      weight_cast += (output_size_quarter * (warp_num << 3));
      int iterations = hidden_dim / (WARP_SIZE << 3) - 1;
      for (int u = 0; u < iterations; u++) {
        if (col < output_size) {
          float4 w_q[2];
#pragma unroll
          for (int m = 0; m < 2; m++) {
            w_q[m] = weight_q[m];
            weight_q[m] = weight_cast[m * output_size];
          }

          __half* inp_data = (__half*)(&val_h);
          int8_t* weight_8 = reinterpret_cast<int8_t*>(w_q);
#pragma unroll
          for (int li = 0; li < 8; li++) {
            float inp_f = inp_data[li];
            sum[0].x += inp_f * weight_8[0];
            sum[0].y += inp_f * weight_8[1];
            sum[1].x += inp_f * weight_8[2];
            sum[1].y += inp_f * weight_8[3];
            weight_8 += 4;
          }
        }
        val_h = vals_cast[gid + (u << 5)];
        weight_cast += (output_size_quarter * (warp_num << 3));
      }
      __half* inp_data = (__half*)(&val_h);
      int8_t* weight_8 = reinterpret_cast<int8_t*>(weight_q);
#pragma unroll
      for (int li = 0; li < 8; li++) {
        float inp_f = inp_data[li];
        sum[0].x += inp_f * weight_8[0];
        sum[0].y += inp_f * weight_8[1];
        sum[1].x += inp_f * weight_8[2];
        sum[1].y += inp_f * weight_8[3];
        weight_8 += 4;
      }
      // quantization scaling
      {
        unsigned q_index = (gid << 2) + (col << 2) * hidden_dim;
        unsigned new_index = q_index / quantization_stride;
        float2 t_scale = shared_quantize_scale[new_index];
        float* scale_f = (float*)&t_scale;
        sum[0].x *= scale_f[0];
        sum[0].y *=
            scale_f[((q_index + hidden_dim) / quantization_stride) - new_index];
        sum[1].x *= scale_f[((q_index + hidden_dim * 2) / quantization_stride) -
                            new_index];
        sum[1].y *= scale_f[((q_index + hidden_dim * 3) / quantization_stride) -
                            new_index];
      }
    }
    {
      const float2* bias_cast;
      if (bias) bias_cast = reinterpret_cast<const float2*>(bias);

      {
        partial_result[gid * (WARP_SIZE + 1) + lane] = sum[0];
        partial_result[(gid + warp_num) * (WARP_SIZE + 1) + lane] = sum[1];
        __syncthreads();

        sum[0] = partial_result[lane * (WARP_SIZE + 2) + gid];
        sum[1] = partial_result[(lane + warp_num) * (WARP_SIZE + 1) + gid];

#pragma unroll
        for (int i = 1; i < WARP_SIZE; i *= 2) {
          sum[0].x += g.shfl_xor(sum[0].x, i);
          sum[0].y += g.shfl_xor(sum[0].y, i);
          sum[1].x += g.shfl_xor(sum[1].x, i);
          sum[1].y += g.shfl_xor(sum[1].y, i);
        }

        if (lane == 0) {
          partial_result[gid] = sum[0];
          partial_result[gid + WARP_SIZE] = sum[1];
        }
        __syncthreads();

        if (gid == 0) {
          sum[0] = partial_result[lane];
          sum[1] = partial_result[lane + WARP_SIZE];
        }
      }

      if (gid == 0) {
        int col = (blockIdx.x % outputBlocks) * WARP_SIZE + lane;
        if (col < output_size) {
          if (bias && blockIdx.x < outputBlocks) {
            float2 bias_ff = bias_cast[col];
            __half2* bias_h = reinterpret_cast<__half2*>(&bias_ff);
            float2 bias_f[2];
            bias_f[0] = __half22float2(bias_h[0]);
            bias_f[1] = __half22float2(bias_h[1]);
            sum[0].x += bias_f[0].x;
            sum[0].y += bias_f[0].y;
            sum[1].x += bias_f[1].x;
            sum[1].y += bias_f[1].y;
            if (add_gelu && gridDim.x == outputBlocks) {
              sum[0].x = gelu(sum[0].x);
              sum[0].y = gelu(sum[0].y);
              sum[1].x = gelu(sum[1].x);
              sum[1].y = gelu(sum[1].y);
            }
          }
          float2 result;
          __half2* result_h = reinterpret_cast<__half2*>(&result);
          result_h[0] = __float22half2_rn(sum[0]);
          result_h[1] = __float22half2_rn(sum[1]);
          output_cast[col] = result;
        }
      }
    }
  }
  // #endif
}

// https://github.com/microsoft/DeepSpeed-internal/blob/reyazda/fast-attn/csrc/transformer/inference_specialized/csrc/custom_gemm.cu#L714
__global__ void input_tiled_gemm_kernel_v2(
    __half* output, const __half* vals, const __half* weight,
    const __half* bias, __half* block_sums, unsigned int hidden_dim,
    unsigned int block_reduce, unsigned int input_size,
    unsigned int output_size, unsigned int outputBlocks,
    unsigned int blockStride, bool add_gelu = false) {
  // #if __CUDA_ARCH__ >= 700
  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);

  unsigned int gid = threadIdx.x >> 5;
  unsigned int lane = threadIdx.x & 0x1f;

  int warp_num = blockDim.x >> 5;

  __half2* output_cast = reinterpret_cast<__half2*>(
      ((gridDim.x == outputBlocks) ? output : block_sums));
  const __half2* vals_cast = reinterpret_cast<const __half2*>(vals);
  const __half2* weight_cast = reinterpret_cast<const __half2*>(weight);
  output_cast += (unsigned)(blockIdx.x / outputBlocks) * (output_size);
  int hidden_half = hidden_dim >> 1;
  weight_cast += ((unsigned)(blockIdx.x / outputBlocks) * blockStride);
  vals_cast += (unsigned)(blockIdx.x / outputBlocks) * hidden_half;

  for (int j = 0; j < input_size; j += (INPUT_TILE2)) {
    __half2 sum[INPUT_TILE2];
#pragma unroll
    for (int t = 0; t < INPUT_TILE2; t++) {
      sum[t] = __float2half2_rn(0.f);
    }

    {
      int wid = gid << loop_unroll_bits;
      weight_cast +=
          wid * output_size + (blockIdx.x % outputBlocks) * WARP_SIZE + lane;

      while (wid < hidden_dim) {
        __shared__ __half2
            vals_h[(loop_unroll >> 1) * INPUT_TILE2 * MAX_WARP_NUM];
        {
          // we read (loop_unroll >> 2) half-2 values per lane, and for 2 times
          // of the INPUT_TILE this makes more threads engaged in reading data
          // from shared memory into registers!
          if (lane < (INPUT_TILE2 << 1)) {
            if (((lane >> 1) + j) < input_size) {
              // here, we consider loop_unroll is always higher that 4!
              unsigned int inp_id = ((lane % 2) << (loop_unroll_bits - 2));

              unsigned int offset =
                  (j + (lane >> 1)) * (block_reduce * (hidden_dim >> 1)) +
                  inp_id;
#pragma unroll
              for (int li = 0; li < (loop_unroll >> 2); li++) {
                vals_h[li + inp_id + (((lane >> 1) << (loop_unroll_bits - 1))) +
                       (gid << (loop_unroll_bits - 1)) * INPUT_TILE2] =
                    vals_cast[offset + (wid >> 1) + li];
              }
            }
          }
          g.sync();
        }

        int col = (blockIdx.x % outputBlocks) * WARP_SIZE + lane;

        if (col < output_size) {
          __half2 weight_h[loop_unroll];
#pragma unroll
          for (int k = 0; k < loop_unroll; k++)
            weight_h[k] = weight_cast[output_size * k];
          auto internal_offset = (gid << (loop_unroll_bits - 1)) * INPUT_TILE2;
#pragma unroll
          for (int t = 0; t < INPUT_TILE2 && (t + j) < input_size; t++) {
            __half2* base_input =
                vals_h + (t << (loop_unroll_bits - 1)) + internal_offset;
#pragma unroll
            for (int li = 0; li < (loop_unroll >> 1); li++) {
              __half* inp_data = reinterpret_cast<__half*>(base_input + li);
              sum[t] += __halves2half2(inp_data[0], inp_data[0]) *
                        weight_h[(li << 1)];
              sum[t] += __halves2half2(inp_data[1], inp_data[1]) *
                        weight_h[(li << 1) + 1];
            }
          }
        }
        wid += warp_num << loop_unroll_bits;
        weight_cast += (output_size * (warp_num << loop_unroll_bits));
      }
    }
    {
      const __half2* bias_cast;
      if (bias) bias_cast = reinterpret_cast<const __half2*>(bias);
      __shared__ __half2 partial_result[2 * MAX_WARP_NUM * (WARP_SIZE + 2)];

      for (int t = 0; t < INPUT_TILE2; t += 2) {
        if ((t + j) < input_size) {
          partial_result[(gid << 1) * (WARP_SIZE + 2) + (lane << 1)] = sum[t];
          partial_result[(gid << 1) * (WARP_SIZE + 2) + (lane << 1) + 1] =
              sum[t + 1];
          b.sync();

          float2 sum_f[2];
          sum_f[0] = __half22float2(
              partial_result[(lane << 1) * (WARP_SIZE + 2) + (gid << 1)]);
          sum_f[1] = __half22float2(
              partial_result[(lane << 1) * (WARP_SIZE + 2) + (gid << 1) + 1]);

#pragma unroll
          for (int i = 1; i < WARP_SIZE; i *= 2) {
            sum_f[0].x += g.shfl_xor(sum_f[0].x, i);
            sum_f[1].y += g.shfl_xor(sum_f[1].y, i);
            sum_f[1].x += g.shfl_xor(sum_f[1].x, i);
            sum_f[0].y += g.shfl_xor(sum_f[0].y, i);
          }

          if (lane == 0) {
            partial_result[(gid << 1)] = __float22half2_rn(sum_f[0]);
            partial_result[(gid << 1) + 1] = __float22half2_rn(sum_f[1]);
          }
          b.sync();

          if (gid == (t >> 1)) {
            sum[t] = partial_result[(lane << 1)];
            sum[t + 1] = partial_result[(lane << 1) + 1];
          }
        }
      }

      if ((gid << 1) < INPUT_TILE2 && ((gid << 1) + j) < input_size) {
        int col = (blockIdx.x % outputBlocks) * WARP_SIZE + lane;
        if (col < output_size) {
          if (bias && blockIdx.x < outputBlocks) {
            __half2 bias_h = bias_cast[col];
            float2 bias_f = __half22float2(bias_h);
            float2 sum_f[2];
            sum_f[0] = __half22float2(sum[(gid << 1)]);
            sum_f[1] = __half22float2(sum[(gid << 1) + 1]);
            sum_f[0].x += bias_f.x;
            sum_f[0].y += bias_f.y;
            sum_f[1].x += bias_f.x;
            sum_f[1].y += bias_f.y;
            if (add_gelu && gridDim.x == outputBlocks) {
              sum_f[0].x = gelu(sum_f[0].x);
              sum_f[0].y = gelu(sum_f[0].y);
              sum_f[1].x = gelu(sum_f[1].x);
              sum_f[1].y = gelu(sum_f[1].y);
            }
            sum[(gid << 1)] = __float22half2_rn(sum_f[0]);
            sum[(gid << 1) + 1] = __float22half2_rn(sum_f[1]);
          }
          output_cast[col + (j + (gid << 1)) * (block_reduce * output_size)] =
              (sum[(gid << 1)]);
          if (((gid << 1) + j + 1) < input_size)
            output_cast[col +
                        (j + (gid << 1) + 1) * (block_reduce * output_size)] =
                (sum[(gid << 1) + 1]);
        }
      }
    }
    weight_cast = reinterpret_cast<const __half2*>(weight);
    weight_cast += ((blockIdx.x / outputBlocks) * blockStride);
  }
  // #endif
}

// https://github.com/microsoft/DeepSpeed-internal/blob/inference-specialized-only/csrc/transformer/inference_specialized/csrc/custom_gemm.cu#L767
__global__ void block_reduce_kernel(__half* output, __half* block_sums,
                                    unsigned batch, unsigned int output_size,
                                    bool add_gelu = false) {
  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);
  unsigned total_count = batch * output_size;
  unsigned int gid = threadIdx.x >> 5;
  unsigned int lane = threadIdx.x & 0x1f;
  unsigned int warp_num = blockDim.x >> 5;

  __half2* output_cast = reinterpret_cast<__half2*>(output);
  __half2* block_sums_cast = reinterpret_cast<__half2*>(block_sums);

  unsigned int col_index = blockIdx.x * WARP_SIZE + lane;
  block_sums_cast += gid * output_size;

  if (col_index < total_count) {
    __shared__ __half2 data_shared[MAX_WARP_NUM * (WARP_SIZE + 1)];

    data_shared[gid * (WARP_SIZE) + lane] =
        block_sums_cast[(col_index / output_size) * (warp_num * output_size) +
                        col_index % output_size];

    b.sync();

    float2 data = __half22float2(
        data_shared[(lane % warp_num) * WARP_SIZE +
                    gid * (WARP_SIZE / warp_num) + (lane / warp_num)]);

    b.sync();
#pragma unroll
    for (int i = 1; i < warp_num; i <<= 1) {
      data.x += g.shfl_xor(data.x, i);
      data.y += g.shfl_xor(data.y, i);
    }

    if ((lane % warp_num) == 0) {
      if (add_gelu) {
        data.x = gelu(data.x);
        data.y = gelu(data.y);
      }
      data_shared[gid * (WARP_SIZE / warp_num) + (lane / warp_num)] =
          __float22half2_rn(data);
    }

    b.sync();

    if (gid == 0) output_cast[col_index] = data_shared[lane];
  }
}

// https://github.com/microsoft/DeepSpeed-internal/blob/reyazda/fast-attn/csrc/transformer/inference_specialized/csrc/custom_gemm.cu#L982
template <typename T>
void launch_input_tiled_gemm_kernel_v2(
    T* output, const T* vals, const int8_t* weight, const T* bias,
    unsigned int hidden_dim, unsigned int input_size, unsigned int output_size,
    float* scale, unsigned int groups, unsigned int merge_count, T* block_sums,
    bool add_gelu, hipStream_t stream) {
  output_size /= 4;
  int outputBlocks = (output_size - 1) / WARP_SIZE + 1;

  int block_reduce = 2;  //(SMs > outputBlocks ? SMs / outputBlocks : 1);
  int br2 = 1;           //(int)log2(block_reduce);
                         //
  // block_reduce = (int)pow(2.0, (float)br2);

  hidden_dim /= block_reduce;

  constexpr int threads = 1024;
  int blockStride = (output_size >> 2) * hidden_dim;

  dim3 grid_dim(outputBlocks * block_reduce);
  dim3 block_dim(threads);
  hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
  input_tiled_gemm_kernel_v2<<<grid_dim, block_dim, 0, stream>>>(
      output, vals, weight, bias, hidden_dim, br2, input_size, output_size,
      outputBlocks, blockStride, scale, groups, block_sums, merge_count,
      (((hidden_dim << br2) >> (merge_count)) * (output_size << 2)) / groups,
      add_gelu);
  if (block_reduce > 1) {
    output_size <<= 1;
    dim3 grids(((output_size * input_size) - 1) / WARP_SIZE + 1);
    dim3 blocks(block_reduce * WARP_SIZE);
    block_reduce_kernel<<<grids, blocks, 0, stream>>>(
        output, block_sums, input_size, (output_size), add_gelu);
  }
}

// https://github.com/microsoft/DeepSpeed-internal/blob/reyazda/fast-attn/csrc/transformer/inference_specialized/csrc/custom_gemm.cu#L1094
template <typename T>
void launch_input_tiled_gemm_kernel_v2(T* output, const T* vals,
                                       const T* weight, const T* bias,
                                       T* block_sums, unsigned int hidden_dim,
                                       unsigned int input_size,
                                       unsigned int output_size, bool add_gelu,
                                       hipStream_t stream) {
  output_size /= 2;
  int outputBlocks = (output_size - 1) / WARP_SIZE + 1;

  int block_reduce = (SMs > outputBlocks ? SMs / outputBlocks : 1);
  int br2 = (int)log2(block_reduce);
  block_reduce = (int)pow(2.0, (float)br2);

  constexpr int threads = 1024;
  int blockStride = (output_size * hidden_dim) / block_reduce;

  dim3 grid_dim(outputBlocks * block_reduce);
  dim3 block_dim(threads);
  input_tiled_gemm_kernel_v2<<<grid_dim, block_dim, 0, stream>>>(
      output, vals, weight, bias, block_sums, hidden_dim / block_reduce,
      block_reduce, input_size, output_size, outputBlocks, blockStride,
      add_gelu);
  if (block_reduce > 1) {
    dim3 grids(((output_size * input_size) - 1) / WARP_SIZE + 1);
    dim3 blocks(block_reduce * WARP_SIZE);
    block_reduce_kernel<<<grids, blocks, 0, stream>>>(
        output, block_sums, input_size, (output_size), add_gelu);
  }
}

template <typename T>
void allocat_workspace(unsigned hidden_dim, unsigned max_seq_len,
                       unsigned batch_size, unsigned head_size = 128) {
  size_t _workSpaceSize = 3 * (hidden_dim * batch_size * max_seq_len);
  Context::Instance().GenWorkSpace(_workSpaceSize, sizeof(T));
}

// int main() {
void run_int8(benchmark::State& state) {
  // https://github.com/microsoft/DeepSpeed-internal/blob/inference-specialized-only/deepspeed/ops/transformer/inference/transformer_inference.py#L289

  auto hidden_size = 5120;
  torch::Tensor input =
      torch::rand({1, 1, hidden_size}, torch::TensorOptions()
                                           .dtype(torch::kFloat16)
                                           .layout(torch::kStrided)
                                           .device(torch::kCUDA));
  torch::Tensor weight =
      torch::rand({hidden_size, 4 * hidden_size}, torch::TensorOptions()
                                                      .dtype(torch::kFloat16)
                                                      .layout(torch::kStrided)
                                                      .device(torch::kCUDA)
                                                      .requires_grad(true));

  torch::Tensor q_scale = torch::rand({1}, torch::TensorOptions()
                                               .dtype(torch::kFloat32)
                                               .layout(torch::kStrided)
                                               .device(torch::kCUDA));

  int groups = 1;
  int merge_count = 0;

  auto input_cont = input.contiguous();
  auto options = torch::TensorOptions()
                     .dtype(input_cont.options().dtype())
                     .layout(torch::kStrided)
                     .device(torch::kCUDA)
                     .requires_grad(false);

  int bsz = input.size(0) * input.size(1);

  using T = __half;

  auto workspace = Context::Instance().GetWorkSpace();
  if (!workspace) {
    allocat_workspace<T>(input.size(2), 256, input.size(0));
    workspace = Context::Instance().GetWorkSpace();
  }
  auto output = torch::from_blob(
      workspace, {input.size(0), input.size(1), weight.size(1)}, options);

  size_t buff_size = Context::Instance().get_workspace_size() / 3;

  // computing the blocking across K dimension
  int out_blocks = (weight.size(1) - 1) / CACHLINE + 1;
  out_blocks = (out_blocks < SMs) ? (SMs / out_blocks) : 1;
  int br2 = (int)log2(out_blocks);
  out_blocks = (int)pow(2.0, (float)br2);

  auto block_sums = torch::empty(
      {input_cont.size(0) * out_blocks, input_cont.size(1), weight.size(1)},
      options);

  torch::Tensor bias =
      torch::rand({1, hidden_size}, torch::TensorOptions()
                                        .dtype(torch::kFloat16)
                                        .layout(torch::kStrided)
                                        .device(torch::kCUDA));

  int cnt = 30;
  float total = 0;
  for (int i = 0; i < cnt; i++) {
    hipEvent_t startEvent, endEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&endEvent);

    CUDA_CHECK(hipEventRecord(startEvent, 0));

    // https://github.com/microsoft/DeepSpeed-internal/blob/reyazda/fast-attn/csrc/transformer/inference_specialized/csrc/pt_binding.cpp#L688
    launch_input_tiled_gemm_kernel_v2(
        (T*)output.data_ptr(), (T*)input.data_ptr(), (int8_t*)weight.data_ptr(),
        (T*)nullptr, input.size(2), bsz, weight.size(1),
        (float*)q_scale.data_ptr(), groups, merge_count,
        (T*)(workspace + buff_size), false,
        Context::Instance().GetCurrentStream());
    CUDA_CHECK_ERROR();
    CUDA_CHECK(hipEventRecord(endEvent, 0));
    CUDA_CHECK(hipEventSynchronize(endEvent));

    float runtime_ms = 0;
    hipEventElapsedTime(&runtime_ms, startEvent, endEvent);
    // state.SetIterationTime(runtime_ms / 10.0e3);
    std::cout << "runtime_ms = " << runtime_ms << " ms\n";
    if (i != 1) {
      total += runtime_ms;
    }
  }
  std::cout << "average runtime_ms = " << total / (cnt - 1) << " ms\n";
}

int main() {
// void run_fp16(benchmark::State& state) {
  auto hidden_size = 5120;
  torch::Tensor input =
      torch::rand({1, 8, hidden_size}, torch::TensorOptions()
                                           .dtype(torch::kFloat16)
                                           .layout(torch::kStrided)
                                           .device(torch::kCUDA));
  torch::Tensor weight =
      torch::rand({hidden_size, 4 * hidden_size}, torch::TensorOptions()
                                                      .dtype(torch::kFloat16)
                                                      .layout(torch::kStrided)
                                                      .device(torch::kCUDA)
                                                      .requires_grad(true));
  auto options = torch::TensorOptions()
                     .dtype(input.options().dtype())
                     .layout(torch::kStrided)
                     .device(torch::kCUDA)
                     .requires_grad(false);

  using T = __half;

  auto workspace = Context::Instance().GetWorkSpace();
  if (!workspace) {
    allocat_workspace<T>(input.size(2), 256, input.size(0));
    workspace = Context::Instance().GetWorkSpace();
  }
  auto output = torch::from_blob(
      workspace, {input.size(0), input.size(1), weight.size(1)}, options);

  size_t buff_size = Context::Instance().get_workspace_size() / 3;

  int bsz = input.size(0) * input.size(1);

  int out_blocks = (weight.size(1) - 1) / (CACHLINE >> 1) + 1;
  out_blocks = (out_blocks < SMs) ? (SMs / out_blocks) : 1;
  int br2 = (int)log2(out_blocks);
  out_blocks = (int)pow(2.0, (float)br2);

  auto block_sums = torch::from_blob(
      (workspace + buff_size),
      {input.size(0) * out_blocks, input.size(1), weight.size(1)}, options);

  torch::Tensor bias =
      torch::rand({1, 4 * hidden_size}, torch::TensorOptions()
                                            .dtype(torch::kFloat16)
                                            .layout(torch::kStrided)
                                            .device(torch::kCUDA));

  hipEvent_t startEvent, endEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&endEvent);

  int cnt = 30;
  float total = 0;
  for (int i = 0; i < cnt; i++) {
    CUDA_CHECK(hipEventRecord(startEvent, 0));

    // https://github.com/microsoft/DeepSpeed-internal/blob/reyazda/fast-attn/csrc/transformer/inference_specialized/csrc/pt_binding.cpp#L549

    launch_input_tiled_gemm_kernel_v2(
        (T*)output.data_ptr(), (T*)input.data_ptr(), (T*)weight.data_ptr(),
        (T*)nullptr, (T*)block_sums.data_ptr(), input.size(2), bsz,
        weight.size(1), false, Context::Instance().GetCurrentStream());
    CUDA_CHECK_ERROR();
    CUDA_CHECK(hipEventRecord(endEvent, 0));
    CUDA_CHECK(hipEventSynchronize(endEvent));

    float runtime_ms = 0;
    hipEventElapsedTime(&runtime_ms, startEvent, endEvent);
    // state.SetIterationTime(runtime_ms / 10.0e3);
    std::cout << "runtime_ms = " << runtime_ms << " ms\n";
    if (i != 1) {
      total += runtime_ms;
    }
  }
  std::cout << "average runtime_ms = " << total / (cnt - 1) << " ms\n";
}

// BENCHMARK(run_int8)->UseManualTime()->Unit(benchmark::kMillisecond);

// BENCHMARK_MAIN();